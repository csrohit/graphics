#include "hip/hip_runtime.h"

#include <cstdint>
#include <iostream>
#define USE_MATH_DEFINES
#include <GL/gl.h>
#include <GL/glu.h>
#include <GL/glx.h>
#include <X11/X.h>
#include <X11/XKBlib.h>
#include <X11/Xlib.h>
#include <X11/Xutil.h>
#include <X11/keysymdef.h>
#include <cstdio>
#include <cstdlib>
#include <math.h>
#define WIN_WIDTH  800
#define WIN_HEIGHT 600

void        HSBtoRGB(double hue, double saturation, double brightness, GLubyte *out);
Display    *dpy              = nullptr; // connection to x-server
Window      w                = 0UL;     // handle to current window
GLXContext  glCtxt           = nullptr; // pointer to opengl context
bool        gbAbortFlag      = false;
bool        gbFullscreen     = false;
static Atom wm_delete_window = 0; // atomic variable to detect close button click

XRectangle rect                 = {0}; // window dimentions rectangle
GLuint     uTextureCheckerBoard = 0U;

#define WIDTH    2400  // Image width
#define HEIGHT   1800  // Image height
#define MAX_ITER 1000  // Maximum number of iterations for each pixel
#define CXMIN    -2.0f // Minimum real value for the subset
#define CXMAX    2.0f  // Maximum real value for the subset
#define CYMIN    -1.5f // Minimum imaginary value for the subset
#define CYMAX    2.5f  // Maximum imaginary value for the subset
int     output[WIDTH][HEIGHT]        = {};
GLubyte checkImage[WIDTH][HEIGHT][4] = {0}; // OpenGL is column major

float cx = -0.7269;// -0.8f; //0.0f;
    float cy = 0.1889; //0.156f; //0.0f;

int  *dev_output;

typedef struct
{
    uint8_t red;
    uint8_t green;
    uint8_t blue;
} RGB;

void writePPM(const char *filename)
{
    FILE *file = fopen(filename, "wb");
    if (!file)
    {
        printf("Error opening file!\n");
        return;
    }

    fprintf(file, "P3\n%d %d\n255\n", WIDTH, HEIGHT);
    for (uint32_t idx = 0U; idx < WIDTH; idx++)
    {
        for (uint32_t idy = 0U; idy < HEIGHT; idy++)
        {
            int   val = output[idx][idy];
            if(val < MAX_ITER)
            { 
            float t   = ((float)val * 360.0f) / MAX_ITER;
            HSBtoRGB(t, 1.0, 1.0, checkImage[idx][idy]);
            }
            else {
                checkImage[idx][idy][0] = 0;
                checkImage[idx][idy][1] = 0;
                checkImage[idx][idy][2] = 0;
            }
            fprintf(file, "%d %d %d\n", checkImage[idx][idy][0], checkImage[idx][idy][1], checkImage[idx][idy][2]);
        }
    }

    fclose(file);
}
void HSBtoRGB(double hue, double saturation, double brightness, GLubyte *out)
{
    hue        = fmod(hue, 360.0);                                                   // Ensure hue is within [0, 360) degrees
    saturation = (saturation > 1.0) ? 1.0 : ((saturation < 0.0) ? 0.0 : saturation); // Limit saturation within [0, 1]
    brightness = (brightness > 1.0) ? 1.0 : ((brightness < 0.0) ? 0.0 : brightness); // Limit brightness within [0, 1]

    double c = saturation * brightness;
    double x = c * (1 - fabs(fmod(hue / 60.0, 2) - 1));
    double m = brightness - c;

    double r1, g1, b1;

    if (hue >= 0 && hue < 60)
    {
        r1 = c;
        g1 = x;
        b1 = 0;
    }
    else if (hue >= 60 && hue < 120)
    {
        r1 = x;
        g1 = c;
        b1 = 0;
    }
    else if (hue >= 120 && hue < 180)
    {
        r1 = 0;
        g1 = c;
        b1 = x;
    }
    else if (hue >= 180 && hue < 240)
    {
        r1 = 0;
        g1 = x;
        b1 = c;
    }
    else if (hue >= 240 && hue < 300)
    {
        r1 = x;
        g1 = 0;
        b1 = c;
    }
    else
    {
        r1 = c;
        g1 = 0;
        b1 = x;
    }

    out[0] = (uint8_t)((r1 + m) * 255);
    out[1] = (uint8_t)((g1 + m) * 255);
    out[2] = (uint8_t)((b1 + m) * 255);
}
__global__ void mandelbrot(float cx, float cy, int *output)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < HEIGHT && col < WIDTH)
    {
        float    zx = CXMIN + (CXMAX - CXMIN) * col / WIDTH;
        float    zy = CYMIN + (CYMAX - CYMIN) * row / WIDTH;
        uint32_t n  = 0;
        for (n = 0; n < MAX_ITER; n++)
        {
            if ((zx * zx + zy * zy) >= 4.0f)
            {
                break;
            }
            float temp = zx * zx - zy * zy + cx;
            zy         = 2 * zx * zy + cy;
            zx         = temp;
            n++;
        }
        output[row * WIDTH + col] = n;
    }
}
static void toggleFullscreen(Display *display, Window window)
{
    XEvent xev;

    Atom wm_state   = XInternAtom(display, "_NET_WM_STATE", False);
    Atom fullscreen = XInternAtom(display, "_NET_WM_STATE_FULLSCREEN", False);

    xev.xclient.type         = ClientMessage;
    xev.xclient.serial       = 0;
    xev.xclient.send_event   = True;
    xev.xclient.message_type = wm_state;
    xev.xclient.format       = 32;
    xev.xclient.window       = window;
    xev.xclient.data.l[0]    = 2; // _NET_WM_STATE_TOGGLE
    xev.xclient.data.l[1]    = fullscreen;
    xev.xclient.data.l[2]    = 0;

    XSendEvent(display, DefaultRootWindow(display), False, SubstructureRedirectMask | SubstructureNotifyMask, &xev);
}
void makeCheckImage(void)
{
    // Define grid and block dimensions for kernel execution
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((WIDTH + threadsPerBlock.x - 1) / threadsPerBlock.x, (HEIGHT + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Launch kernel
    mandelbrot<<<numBlocks, threadsPerBlock>>>(cx, cy, dev_output);

    // Copy result from device to host
    hipMemcpy(output, dev_output, WIDTH * HEIGHT * sizeof(int), hipMemcpyDeviceToHost);

    // Write computed Mandelbrot set to a PPM file
    for (uint32_t idx = 0U; idx < WIDTH; idx++)
    {
        for (uint32_t idy = 0U; idy < HEIGHT; idy++)
        {
            
            int   val = output[idx][idy];
            if(val < MAX_ITER)
            {
                float t   = ((float)val / MAX_ITER) * 460.0f;
                HSBtoRGB(t, 1.0, 1.0, checkImage[idx][idy]);
            }
            else {
                checkImage[idx][idy][0] = 0;
                checkImage[idx][idy][1] = 0;
                checkImage[idx][idy][2] = 0;
            }

        }
    }
    // Free device and host memory
}
void loadGLTexture1(void)
{
    /* Local variable declarations */
    makeCheckImage();

    /* Bind Generated texture */
    glBindTexture(GL_TEXTURE_2D, uTextureCheckerBoard);

    // alignment and unpacking
    glPixelStorei(GL_UNPACK_ALIGNMENT, 1);

    /* Set Texture parameters
        Nearest - Better for processing and comparatively poor on quality
        Linear - Poor in processing and better on quality
    */
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);

    /* build mipmap images */
    // gluBuild2DMipmaps(GL_TEXTURE_2D, 3, bmp.bmWidth, bmp.bmHeight, GL_BGR_EXT, GL_UNSIGNED_BYTE, (void *)bmp.bmBits);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, WIDTH, HEIGHT, 0, GL_RGBA, GL_UNSIGNED_BYTE, (void *)checkImage);

    /* Unbind texture */
    glBindTexture(GL_TEXTURE_2D, 0);
}

void createWindow()
{
    dpy = XOpenDisplay(nullptr);

    if (dpy == nullptr)
    {
        fprintf(stderr, "Error: Could not open X display\n");
        exit(1);
    }

    int screen = DefaultScreen(dpy);

    // Define the attributes for the visual context
    static int visualAttributes[] = {
        GLX_RGBA, GLX_DOUBLEBUFFER, GLX_RGBA, GLX_DOUBLEBUFFER, GLX_DEPTH_SIZE, 24, GLX_STENCIL_SIZE, 8, GLX_RED_SIZE, 8, GLX_GREEN_SIZE, 8, GLX_BLUE_SIZE, 8, GLX_SAMPLE_BUFFERS, 0, GLX_SAMPLES, 0,
        None};

    XVisualInfo *visual = glXChooseVisual(dpy, screen, visualAttributes);
    if (visual == nullptr)
    {
        fprintf(stderr, "Error: No appropriate visual found\n");
        exit(1);
    }

    Colormap             colormap = XCreateColormap(dpy, RootWindow(dpy, screen), visual->visual, AllocNone);
    XSetWindowAttributes windowAttributes;
    windowAttributes.colormap   = colormap;
    windowAttributes.event_mask = ExposureMask | KeyPressMask | StructureNotifyMask;

    w = XCreateWindow(dpy, RootWindow(dpy, screen), 0, 0, WIN_WIDTH, WIN_HEIGHT, 0, visual->depth, InputOutput, visual->visual, CWColormap | CWEventMask, &windowAttributes);

    XMapWindow(dpy, w);
    XStoreName(dpy, w, "OpenGL Triangle");
    toggleFullscreen(dpy, w);

    glCtxt = glXCreateContext(dpy, visual, nullptr, GL_TRUE);
    glXMakeCurrent(dpy, w, glCtxt);

    // Allocate memory for output
    glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
    hipMalloc((void **)&dev_output, WIDTH * HEIGHT * sizeof(int));
    // Allocate memory on the GPU
    /* Create OpenGL Texture Object */
    glGenTextures(1, &uTextureCheckerBoard);
    // loading images to create texture

    loadGLTexture1();
    /* Enable texture */
    glEnable(GL_TEXTURE_2D);

    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
}

void renderScene()
{
    // code
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT); // clear the window with color whose bit is set
    glMatrixMode(GL_MODELVIEW);                         // for matrix calculation while displaying use GL_MODELVIEW beacuse need to display something
    glLoadIdentity();                                   // take identity matrix for beginning

    /* Do initial tranlation for better visibility */
    // glTranslatef(0.0f, 0.0f, -4.0f);

    glBindTexture(GL_TEXTURE_2D, uTextureCheckerBoard);
    glBegin(GL_QUADS);

    /* Top-Right */
    glTexCoord2f(1.0f, 1.0f);
    glVertex3f(1.0f, 1.0f, 0.0f);

    /* Top-Left */
    glTexCoord2f(0.0f, 1.0f);
    glVertex3f(-1.0f, 1.0f, 0.0f);

    /* Bottom-Left */
    glTexCoord2f(0.0f, 0.0f);
    glVertex3f(-1.0f, -1.0f, 0.0f);

    /* Bottom-Right */
    glTexCoord2f(1.0f, 0.0f);
    glVertex3f(1.0f, -1.0f, 0.0f);

    glEnd();
    glBindTexture(GL_TEXTURE_2D, 0);
}
void resize(int width, int height)
{
    if (height <= 0)
        height = 1;

    glMatrixMode(GL_PROJECTION); // for matrix calculation while resizing use GL_PROJECTION
    glLoadIdentity();            // take identity matrix for beginning

    GLdouble gldHeight = (GLdouble)(tan(M_PI / 8.0f) * 0.1);
    GLdouble gldWidth  = gldHeight * ((GLdouble)width / (GLdouble)height);

    /* after setting projection matrix, ensure that VIEW matrix is also set */
    // glFrustum(-gldWidth, gldWidth, -gldHeight, gldHeight, 0.1, 100.0);
    // gluLookAt(0.0, 0.0, 3.0, 0.0, 0.0, 0.0, 0.0, 1.0, 0.0);
    glViewport(0, 0, width, height); // view complete window
}

void update()
{
}

int main(int argc, char *argv[])
{
    createWindow();
    bool shouldDraw = false;
    shouldDraw      = false;
    while (!gbAbortFlag)
    {
        XEvent event;
        if (XPending(dpy))
        {
            XNextEvent(dpy, &event);
            switch (event.type)
            {
                case Expose:
                {
                    if (!shouldDraw)
                        shouldDraw = true;
                    break;
                }
                case ClientMessage:
                {
                    if (event.xclient.data.l[0] == wm_delete_window)
                    {
                        gbAbortFlag = true;
                    }

                    break;
                }
                case KeyPress:
                {
                    KeySym sym = XkbKeycodeToKeysym(dpy, event.xkey.keycode, 0, 0);

                    switch (sym)
                    {
                        case XK_a:
                        {
                            writePPM("key.ppm");
                            if (event.xkey.state & ShiftMask)
                            {
                                /* handle A */
                            }
                            else
                            {
                            }
                            break;
                        }
                        case XK_f:
                        {
                            toggleFullscreen(dpy, w);
                            gbFullscreen = !gbFullscreen;
                            break;
                        }
                        case XK_r:
                        {
                            break;
                        }
                        case XK_Escape:
                        {
                            gbAbortFlag = true;
                            break;
                        }
                    }
                    break;
                }
                case ConfigureNotify:
                {
                    if (rect.width != event.xconfigure.width || rect.height != event.xconfigure.height)
                    {
                        resize(event.xconfigure.width, event.xconfigure.height);
                        rect.width  = event.xconfigure.width;
                        rect.height = event.xconfigure.height;
                    }
                    break;
                }
            }
        }

        if (!shouldDraw)
            continue;
        update();
        renderScene();

        glXSwapBuffers(dpy, w);
    }
    hipFree(dev_output);
    glDeleteTextures(1, &uTextureCheckerBoard);
    glXMakeCurrent(dpy, None, nullptr);
    glXDestroyContext(dpy, glCtxt);
    XDestroyWindow(dpy, w);
    XCloseDisplay(dpy);

    return 0;
}
