
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>

#define WIDTH 2400 // Image width
#define HEIGHT 1800 // Image height
#define MAX_ITER 1000 // Maximum number of iterations for each pixel
#define CXMIN -2.0f // Minimum real value for the subset
#define CXMAX 2.0f // Maximum real value for the subset
#define CYMIN -1.5f // Minimum imaginary value for the subset
#define CYMAX 1.5f // Maximum imaginary value for the subset


typedef struct
{
    uint8_t red;
    uint8_t green;
    uint8_t blue;
} RGB;

RGB HSBtoRGB(double hue, double saturation, double brightness) {
    hue = fmod(hue, 360.0); // Ensure hue is within [0, 360) degrees
    saturation = (saturation > 1.0) ? 1.0 : ((saturation < 0.0) ? 0.0 : saturation); // Limit saturation within [0, 1]
    brightness = (brightness > 1.0) ? 1.0 : ((brightness < 0.0) ? 0.0 : brightness); // Limit brightness within [0, 1]

    double c = saturation * brightness;
    double x = c * (1 - fabs(fmod(hue / 60.0, 2) - 1));
    double m = brightness - c;

    double r1, g1, b1;

    if (hue >= 0 && hue < 60) {
        r1 = c;
        g1 = x;
        b1 = 0;
    } else if (hue >= 60 && hue < 120) {
        r1 = x;
        g1 = c;
        b1 = 0;
    } else if (hue >= 120 && hue < 180) {
        r1 = 0;
        g1 = c;
        b1 = x;
    } else if (hue >= 180 && hue < 240) {
        r1 = 0;
        g1 = x;
        b1 = c;
    } else if (hue >= 240 && hue < 300) {
        r1 = x;
        g1 = 0;
        b1 = c;
    } else {
        r1 = c;
        g1 = 0;
        b1 = x;
    }

    RGB rgb;
    rgb.red = (uint8_t)((r1 + m) * 255);
    rgb.green = (uint8_t)((g1 + m) * 255);
    rgb.blue = (uint8_t)((b1 + m) * 255);

    return rgb;
}
__global__ void mandelbrot(int *output) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < HEIGHT && col < WIDTH) {
        float zx = CXMIN + (CXMAX - CXMIN)*col/WIDTH;
        float zy = CYMIN + (CYMAX - CYMIN)*row/WIDTH;
        float cx = -0.7269;// -0.8f; //0.0f;
        float cy = 0.1889; //0.156f; //0.0f;
        uint32_t n = 0;
        for (n = 0; n < MAX_ITER; n++)
        {
            if ((zx * zx + zy * zy) >= 4.0f)
            {
                break;
            }
            float temp = zx * zx - zy * zy + cx;
            zy = 2 * zx * zy + cy;
            zx = temp;
            n++;
        }
        output[row * WIDTH + col] = n;
    }
}

void writePPM(const char *filename, int *data) {
    FILE *file = fopen(filename, "wb");
    if (!file) {
        printf("Error opening file!\n");
        return;
    }

    fprintf(file, "P3\n%d %d\n255\n", WIDTH, HEIGHT);
    for (int i = 0; i < WIDTH * HEIGHT; ++i) {
        int val = data[i];

            RGB color;
        if (val < MAX_ITER) {
            float t = ((float)val *360.0f)/ MAX_ITER;
            color = HSBtoRGB(t, 1.0f, 1.0f);
        }
        else
        {
            color = {};
        }

        fprintf(file, "%d %d %d\n", color.red, color.green, color.blue);
    }

    fclose(file);
}

int main() {
    int *output, *dev_output;

    // Allocate memory for output
    output = (int *)malloc(WIDTH * HEIGHT * sizeof(int));

    // Allocate memory on the GPU
    hipMalloc((void **)&dev_output, WIDTH * HEIGHT * sizeof(int));

    // Define grid and block dimensions for kernel execution
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((WIDTH + threadsPerBlock.x - 1) / threadsPerBlock.x, (HEIGHT + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Launch kernel
    mandelbrot<<<numBlocks, threadsPerBlock>>>(dev_output);

    // Copy result from device to host
    hipMemcpy(output, dev_output, WIDTH * HEIGHT * sizeof(int), hipMemcpyDeviceToHost);

    // Write computed Mandelbrot set to a PPM file
    writePPM("mandelbrot_color.ppm", output);

    // Free device and host memory
    hipFree(dev_output);
    free(output);

    return 0;
}

